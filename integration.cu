#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__device__ double f(double x) {
    return x * x;
}

__global__ void trapezoidalKernel(double a, double h, int n, double *partial_sum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < n-1) {
        double x = a + idx * h;
        double fx = f(x);
        double fx_next = f(x + h);
        partial_sum[idx] = h * (fx + fx_next) / 2.0;
    }
}

double cudaIntegrate(double a, double b, int n) {
    double h = (b - a) / n; 
    double *d_partial_sum;
    hipMalloc(&d_partial_sum, (n-1) * sizeof(double));
    
    int blockSize = 256;
    int gridSize = (n-1 + blockSize - 1) / blockSize;
    
    trapezoidalKernel<<<gridSize, blockSize>>>(a, h, n, d_partial_sum);  
    hipDeviceSynchronize();
    
    double *h_partial_sum = (double*)malloc((n-1) * sizeof(double));
    hipMemcpy(h_partial_sum, d_partial_sum, (n-1) * sizeof(double), hipMemcpyDeviceToHost);
    
    double sum = 0.0;
    for (int i = 0; i < n-1; i++) {
        sum += h_partial_sum[i];
    }
    
    hipFree(d_partial_sum);
    free(h_partial_sum);
    
    return sum;
}

int main() {
    printf("CUDA Parallel Integration using Trapezoidal Method\n");
    
    // Test Case 1
    double a = 0.0, b = 1.0;
    int n = 1000;
    
    printf("Test Case 1:\n");
    printf("Function: f(x) = x^2\n");
    printf("Interval: [%.1f, %.1f]\n", a, b);
    printf("Number of intervals: %d\n", n);
    
    double analytical = (b*b*b - a*a*a) / 3.0;
    printf("Analytical result: %.10f\n", analytical);
    
    double cuda_result = cudaIntegrate(a, b, n);
    printf("CUDA result: %.10f\n", cuda_result);
    printf("Error: %.10f\n\n", fabs(cuda_result - analytical));
    
    // Test Case 2
    a = 1.0; b = 3.0; n = 5000;
    
    printf("Test Case 2:\n");
    printf("Function: f(x) = x^2\n");
    printf("Interval: [%.1f, %.1f]\n", a, b);
    printf("Number of intervals: %d\n", n);
    
    analytical = (b*b*b - a*a*a) / 3.0;
    printf("Analytical result: %.10f\n", analytical);
    
    cuda_result = cudaIntegrate(a, b, n);
    printf("CUDA result: %.10f\n", cuda_result);
    printf("Error: %.10f\n", fabs(cuda_result - analytical));
    
    return 0;
}
